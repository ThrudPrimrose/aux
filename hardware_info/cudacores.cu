#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int deviceID = 0; // Use 0 if you have a single GPU
    hipSetDevice(deviceID);

    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, deviceID);

    int coresPerSM;
    hipDeviceGetAttribute(&coresPerSM, cudaDevAttrCudaCoresPerMultiprocessor, deviceID);

    int totalCudaCores = numSMs * coresPerSM;

    printf("Total CUDA Cores: %d\n", totalCudaCores);

    return 0;
}