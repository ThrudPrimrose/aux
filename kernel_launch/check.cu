
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void check()
{
    printf("Good\n");
}


void entry()
{
    printf("Entry\n");
    check<<<1,1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
}