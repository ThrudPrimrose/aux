#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include "macros2.h"

__global__ void kernel(){
    printf("Hello from GPU\n");
}

void entry(){
    kernel<<<1,1>>>();
    put();
    hipDeviceSynchronize();
}

int main(){
    entry();
}