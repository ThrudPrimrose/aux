
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int* a, int* b, int* result, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        result[tid] = a[tid] + b[tid];
    }
}

__global__ void stridedVectorAdd(int* a, int* b, int* result, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size / 2) {
        result[tid*2] = a[tid*2] + b[tid*2];
    }
}

int main() {
    int size = 100000000;
    int* a, * b, * result;
    int* d_a, * d_b, * d_result;

    // Allocate memory on host
    a = (int*)malloc(size * sizeof(int));
    b = (int*)malloc(size * sizeof(int));
    result = (int*)malloc(size * sizeof(int));

    // Initialize vectors to zero
    memset(a, 0, size * sizeof(int));
    memset(b, 0, size * sizeof(int));

    // Allocate memory on device
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_result, size * sizeof(int));

    // Copy vectors from host to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Create CUDA events for start and end
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // Record the start event
    hipEventRecord(start);
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, size);
    // Record the end event
    hipEventRecord(end);
    hipEventSynchronize(end);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);

    // Print the elapsed time
    printf("Elapsed time for unit-stride add: %.5f ms\n", milliseconds);

    // Copy result from device to host
    hipMemcpy(result, d_result, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);
    hipMalloc((void**)&d_result, size * sizeof(int));
    
    // Create CUDA events for start and end
    hipEvent_t start2, end2;
    hipEventCreate(&start2);
    hipEventCreate(&end2);

    // Record the start event
    hipEventRecord(start2);
    // Launch kernel
    threadsPerBlock = 256;
    blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    stridedVectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, size);
    
    // Record the end event
    hipEventRecord(end);
    hipEventSynchronize(end);

    // Calculate the elapsed time
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start2, end2);

    // Copy result from device to host
    hipMemcpy(result, d_result, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the elapsed time
    printf("Elapsed time for stridedAdd: %.5f ms\n", milliseconds);

    // Free memory on host and device
    free(a);
    free(b);
    free(result);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
